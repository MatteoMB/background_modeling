#include "hip/hip_runtime.h"
#include <glob.h>
#include <string.h>
#include <iostream>
#include <omp.h>
#define cimg_use_jpeg
#include "CImg.h"
using namespace cimg_library;
using namespace std;
#define RIDX 1
#define RIDY 1
#define D_DIM 8
// #define STOP 30
#define D_HIST 256/(RIDX * RIDY)
#define PATCHX 16 / RIDX
#define PATCHY 16 / RIDY
#define INIT 4
#define SIM 135 / (RIDX * RIDY)
#define A_DUR 75
#define PERFORMANCE
#ifdef PERFORMANCE
    #include <ctime>
    #include <chrono>
#endif

__global__
typedef struct Dict {
    unsigned int D[D_DIM][D_HIST];
    unsigned int C[D_DIM];
    unsigned int A[A_DUR][D_HIST];
    unsigned int iHist = 0;
    unsigned int lenD = 0;
    unsigned int lenA = 0;
} Dict_t;

__global__ 
void kernel(unsigned char * frame, Dict_t * Dictionary, int h, int w){
    //global indexes
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    //local index
    unsigned char k = threadIdx.y * blockDim.x + threadIdx.x;
    //dictionary index
    int l = w / blockDim.x * blockIdx.y + blockIdx.x;
    //iD is the cycling index of the Dictionary structure, sim stands for similarity
    //anomaly means if it's been detected an anomaly in the current patch
    //all the threads in the group must agree on these three variables
    __shared__ unsigned int iD, sim;
    __shared__ bool anomaly;
    if(!k){
        sim = 0;
        iD = 0;
        //by default we assume there's an anomaly
        anomaly = 1;
    }
    //p is the current pixel value
    //each thread works on one pixel (one patch => one thread group)
    unsigned char p = frame[i * w + j];
    int iHist = Dictionary[l].iHist;
    int lenD =  Dictionary[l].lenD;
    int lenA = Dictionary[l].lenA;
    int value, i_a, i_c, min, iMin, avg;
    //here we parallelize the initialization of the current histogram
    //(stored by default in the dictionary at index iHist) 
    Dictionary[l].D[iHist][k] = 0;
    __syncthreads();
    //building the histogram means only incrementing by 1 the corresponding bin
    atomicInc(&Dictionary[l].D[iHist][p],257);
    __syncthreads();
    //for the first INIT images, we just add their histograms to the Dictionary
    if(iHist < INIT){
        if(!k){
            Dictionary[l].C[iHist] = 0;
            Dictionary[l].iHist++;
            Dictionary[l].lenD++;
        }
        return;
    }
    //loop inside the dictionary, we want to compare the current histogram
    //with all the others in the dictionary
    for(; iD < lenD;){
        //The current histogram can be anywhere in the dictionary, we don't compare the histogram with itself
        if(iD != iHist){
            value = Dictionary[l].D[iHist][k];
            //the similarity is the intersection (minimum) between the corresponding bins of the two histograms
            atomicAdd(&sim,value < Dictionary[l].D[iD][k] ? value : Dictionary[l].D[iD][k]);
            __syncthreads();
            //if they are enough similar (>50%), all threads agree that there isn't an anomaly
            if(sim > SIM){
                anomaly = 0;
                //thread 0 register what histogram was used to reconstruct the patch
                if(!k) Dictionary[l].C[iD]++;
                break;
            } 
        }
        //iD++
        if(!k){
            sim = 0;
            atomicInc(&iD, D_DIM + 1);
        }
        __syncthreads();
    }
    if(anomaly){
        //each thread darkens its own pixel
        frame[i*w +j]= p/5;
        //copy current histogram in the container of the anomalies
        Dictionary[l].A[lenA][k] = value;
        __syncthreads();
        //the container of the anomalies is full: there is a change in the background, so we
        //add the average of the histograms of A to the dictionary. It will represent the new background
        if (lenA == A_DUR - 1){
            //each thread averages one bin
            for( i_a = 0, avg = 0; i_a < A_DUR; i_a ++)
                avg += Dictionary[l].A[i_a][k];
            //the average is rounded by excess since we take the minimum in the comparison
            Dictionary[l].D[iHist][k] = avg ? avg / A_DUR + 1 : 0;
            //initialize the Counter
            Dictionary[l].C[iHist] = 0;
                //the aim of this part is to substitute the least used histogram with the new one
                //(since the dictionary is full). It's difficult to parallelize, but the dictionary is
                //small (and almost never filled), so it's done sequentially
            if(!k){
                if(lenD == D_DIM-1){
                    //take the index of the minimum
                    for (i_c = 0, min = 0, iMin = -1; i_c < lenD; i_c++)
                        if(i_c != iHist && (min > Dictionary[l].C[i_c] || iMin < 0)){
                            min = Dictionary[l].C[i_c];
                            iMin = i_c;
                        }
                    //in the next loop,, the iMin-th histogram will be overwritten
                    Dictionary[l].iHist = iMin;
                }
                //there is enough space in the dictionary
                else{
                    Dictionary[l].iHist++;
                    Dictionary[l].lenD++;
                }
                //empty A
                Dictionary[l].lenA = 0;
            }
        }
        //we can take at least one more anomaly
        else if(!k) Dictionary[l].lenA++;
    }
    //no anomaly, background isn't changing: empty A
    else if(!k) Dictionary[l].lenA = 0;
}

int main(int argc, char * argv[]){
    CImg<unsigned char> * imgs;
    CImg<unsigned char> * outImgs;
    unsigned char  **frames, * dFrame;
    bool * flags;
    Dict_t * Dictionary;
    glob_t globbuf;
    int h,w,i,j,iFrame,nFrames;

    if(argc<2){
        cout<<"Missing directory name"<<endl;
        return 1;
    }
    cerr<<"Getting all filenames"<<endl;
    char *dir = strcat(argv[1], "*.jpg");
    if (glob (dir, GLOB_TILDE, NULL, &globbuf) != 0){
            cout << "Can't open the chosen directory" << endl;
            return 1;
    }
    nFrames = globbuf.gl_pathc;
    frames = new unsigned char * [nFrames];
    imgs = new CImg<unsigned char> [nFrames];
    outImgs = new CImg<unsigned char> [nFrames];
    flags = new bool[nFrames];
    for(iFrame = 0; iFrame<nFrames; iFrame++)
        flags[iFrame] = 0;
    #ifdef PERFORMANCE
        chrono::high_resolution_clock::time_point t_start,t_end;
        chrono::duration<double> exec_time;
        double * perf = new double [nFrames];
    #endif
    cerr<<"now starting to read all files"<<endl;
    imgs[0].load(globbuf.gl_pathv[0]);
    h = imgs[0].height();
    w = imgs[0].width();
    #ifdef PERFORMANCE
        t_start = chrono::high_resolution_clock::now();
    #endif
    #pragma omp parallel num_threads(2) shared(flags)
    {
        #pragma omp sections private(iFrame) nowait
        {
            #pragma omp section
            {
                for (iFrame = 0; iFrame < nFrames; iFrame++){
                    if(iFrame)
                        imgs[iFrame].load(globbuf.gl_pathv[iFrame]);
                    if (!iFrame){
                        hipMalloc(&dFrame,sizeof(unsigned char) * h * w);
                        hipMallocManaged(&Dictionary,sizeof(Dict_t) * w/PATCHX * h/PATCHY);
                    }
                    hipHostMalloc(&(frames[iFrame]),sizeof(unsigned char) * h * w);

                    if(imgs[iFrame].height() != h || imgs[iFrame].width()!=w){
                        cerr<<"Frame dimensions don't match"<<endl;
                    }

                    for (i = 0; i < h; i++)
                        for (j = 0; j < w; j++)
                            frames[iFrame][i * w + j] = (unsigned char)(0.299 * imgs[iFrame](j, i, 0, 0) + 0.587 * imgs[iFrame](j, i, 0, 1) + 0.114 * imgs[iFrame](j, i, 0, 2));
                   
                    while(iFrame && !flags[iFrame -1]){
                        #pragma omp flush(flags)
                    }
   
                    dim3 threadsPerBlock(PATCHX, PATCHY);
                    dim3 numBlocks(w / threadsPerBlock.x, h / threadsPerBlock.y);
                    hipMemcpyAsync(dFrame, frames[iFrame], sizeof(unsigned char) * h * w, hipMemcpyHostToDevice);
                    kernel<<<numBlocks, threadsPerBlock>>>(dFrame, Dictionary ,h, w);
                    hipMemcpyAsync(frames[iFrame], dFrame, sizeof(unsigned char) * h * w, hipMemcpyDeviceToHost);
                    imgs[iFrame].assign();
                    flags[iFrame] = 1;
                    #pragma omp flush(flags)
                }  

            }                            
            #pragma omp section 
            {
                for (iFrame = 0; iFrame < nFrames; iFrame++){    
                    while(!flags[iFrame]){
                        #pragma omp flush(flags)
                    };
                    hipDeviceSynchronize();
                    outImgs[iFrame].assign(frames[iFrame],w,h,1,1,true);
                    outImgs[iFrame].save(globbuf.gl_pathv[iFrame],1,3);
                    hipFree(frames[iFrame]);   
                    outImgs[iFrame].assign();     
                }
                

            }
        }
    }
    #ifdef PERFORMANCE
        t_end = chrono::high_resolution_clock::now();
        exec_time = t_end - t_start;
        cout <<"Avg time: "<< exec_time.count() * 1e3 /nFrames <<endl;
    #endif
    free(frames);
    hipFree(dFrame);
    hipFree(Dictionary);
    if (globbuf.gl_pathc > 0)
        globfree(&globbuf);
    return 0;

}